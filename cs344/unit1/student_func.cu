#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  uchar4 rgba = rgbaImage[x * numCols + y];
  greyImage[x * numCols + y] = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
}

int find_devisor(int n) {
  for (int i = 20; i > 0; i--) {
    if (n % i == 0) {
      return i;
     }
  }
  return 1;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  int blockDimX = find_devisor(numRows);
  int blockDimY = find_devisor(numCols);
  const dim3 blockSize(blockDimX, blockDimY, 1);  //TODO
  const dim3 gridSize( numRows / blockDimX, numCols / blockDimY, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}

